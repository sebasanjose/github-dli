#include "hip/hip_runtime.h"
#include <math.h>
#include <time.h>
#include <iostream>
#include <stdexcept>
#include "hip/hip_runtime.h"

// declare the vectors' number of elements and their size in bytes
static const int n_el = 10000000; // 10 millions
static const size_t size = n_el * sizeof(float);

// function for computing sum on CPU
void CPU_sum(const float* A, const float* B, float* C, int n_el) {
  for (int i=0; i<n_el; i++) {
    C[i]=A[i]+B[i];
  }    
}

// kernel
__global__ void kernel_sum(const float* A, const float* B, float* C, int n_el)
{
  // calculate the unique thread index
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  // perform tid-th elements addition 
  if (tid < n_el) C[tid] = A[tid] + B[tid];
}

// function which invokes the kernel
void GPU_sum(const float* A, const float* B, float* C, int n_el) {

  // declare the number of blocks per grid and the number of threads per block
  int threadsPerBlock,blocksPerGrid;

  // use max 512 threads per block
  threadsPerBlock = min(512,n_el);
  blocksPerGrid   = ceil(double(n_el)/double(threadsPerBlock));

  // invoke the kernel
  int numChunks = 24;
  for (int i = 0; i < numChunks; i++) {
    
  }
  kernel_sum<<<blocksPerGrid,threadsPerBlock>>>(A, B, C, n_el);
}

int main(){
  // declare and allocate input vectors h_A and h_B in the host (CPU) memory
  float* h_A = (float*)malloc(size);
  float* h_B = (float*)malloc(size);
  float* h_C = (float*)malloc(size);

  // initialize input vectors
  for (int i=0; i<n_el; i++){
    h_A[i]=sin(i);
    h_B[i]=cos(i);
  }

  /************ CPU Version ***********/

  clock_t tstart,tend;
  float cpu_duration;
  // compute on CPU
  tstart = clock();
  
  /////////////////////////////////
  // call kernel function
  /////////////////////////////////
  CPU_sum(h_A, h_B, h_C, n_el);
  /////////////////////////////////

  tend = clock();
  cpu_duration = ((float)(tend-tstart))/CLOCKS_PER_SEC;
  printf("Total  time for sum on CPU: %f seconds\n",cpu_duration);

  /************ GPU Version ***********/

  clock_t tstart_total;
  tstart_total = clock();

  /////////////////////////////////
  // transfer data from CPU to GPU
  /////////////////////////////////
  // declare device vectors in the device (GPU) memory
  float *d_A,*d_B,*d_C;
  // allocate device vectors in the device (GPU) memory
  hipMalloc(&d_A, size);
  hipMalloc(&d_B, size);
  hipMalloc(&d_C, size);
  // copy input vectors from the host (CPU) memory to the device (GPU) memory
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  float gpu_duration;
  tstart = clock();

  /////////////////////////////////
  // call kernel function
  /////////////////////////////////
  GPU_sum(d_A, d_B, d_C, n_el);
  // wait for everything to finish
  hipDeviceSynchronize();
  /////////////////////////////////

  tend = clock();
  gpu_duration = ((float)(tend-tstart))/CLOCKS_PER_SEC;
  printf("Kernel time for sum on GPU: %f seconds\n",gpu_duration);

  /////////////////////////////////
  // transfer data from GPU to CPU
  /////////////////////////////////
  // copy the output (results) vector from the device (GPU) memory to the host (CPU) memory
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
  // free device memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  // wait for everything to finish
  hipDeviceSynchronize();

  tend = clock();
  gpu_duration = ((float)(tend-tstart_total))/CLOCKS_PER_SEC;
  printf("Total  time for sum on GPU: %f seconds\n",gpu_duration);

  /************ Check correctness using RMS Error ***********/

  // compute the squared error of the result
  // using double precision for good accuracy
  double err=0;
  for (int i=0; i<n_el; i++) {
    double diff=double((h_A[i]+h_B[i])-h_C[i]);
    err+=diff*diff;
    // print results for manual checking.
    //printf("%f=%f,",h_A[i]+h_B[i],h_C[i]);
  }
  // compute the RMS error
  err=sqrt(err/double(n_el));
  printf("error: %f\n",err);

  printf("speed-up: %.2fx",cpu_duration/gpu_duration);

  // free host memory
  free(h_A);
  free(h_B);
  free(h_C);

  return 0;
}